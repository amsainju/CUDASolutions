#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  //TODO
  //Fill in the kernel to convert from color to greyscale
  //the mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //
  //The output (greyImage) at each pixel should be the result of
  //applying the formula: output = .299f * R + .587f * G + .114f * B;
  //Note: We will be ignoring the alpha channel for this conversion

  //First create a mapping from the 2D block and grid locations
  //to an absolute 2D location in the image, they use that to
  //calculate a 1D offset
  int y = threadIdx.y+ blockIdx.y* blockDim.y;
  int x = threadIdx.x+ blockIdx.x* blockDim.x;
  if (y < numCols && x < numRows) {
  	int index = numRows*y +x;
  uchar4 color = rgbaImage[index];
  unsigned char grey = (unsigned char)(0.299f*color.x+ 0.587f*color.y + 0.114f*color.z);
  greyImage[index] = grey;
  }
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  //You must fill in the correct sizes for the blockSize and gridSize
  //currently only one block with one thread is being launched
  
  int   blockWidth = 32;
  
  const dim3 blockSize(blockWidth, blockWidth, 1);
  int   blocksX = numRows/blockWidth+1;
  int   blocksY = numCols/blockWidth+1; //TODO
  const dim3 gridSize( blocksX, blocksY, 1);  //TODO
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}
